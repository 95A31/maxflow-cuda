#include<hip/hip_runtime.h>
#include<bits/stdc++.h>

#define number_of_nodes 6
#define number_of_edges 10
#define threads_per_block 256
#define number_of_blocks_nodes ((number_of_nodes/threads_per_block) + 1)
#define number_of_blocks_edges ((number_of_edges/threads_per_block) + 1)
#define pii std::pair<int,int>
#define KERNEL_CYCLES gpu_graph->V
#define INF 1000000


struct Graph
{
    int V;  // number of vertices
    int excess_total;   // total excess flow over all vertices in the graph
    int *height;    // array containing height values of the vertices
    int *excess_flow;   // array containing excess flow values of the vertices
    pii *adj_mtx;    // array containing the adjacency matrix of the graph as (residual capacity,capacity) pair edges
};

void readgraph(Graph *cpu_graph, int V, int E)
{
    for(int i = 0; i < V; i++)
    {
        for(int j = 0; j < V; j++)
        {
            cpu_graph->adj_mtx[i*V + j].first = 0;
            cpu_graph->adj_mtx[i*V + j].second = 0;
        }
    }

    FILE *fp = fopen("edgelist.txt","r");

    char buf1[10],buf2[10],buf3[10];
    int e1,e2,cp;

    for(int i = 0; i < E; i++)
    {
        fscanf(fp,"%s",buf1);
        fscanf(fp,"%s",buf2);
        fscanf(fp,"%s",buf3);

        e1 = atoi(buf1);
        e2 = atoi(buf2);
        cp = atoi(buf3);

        cpu_graph->adj_mtx[e1*V + e2].first = cp;
        cpu_graph->adj_mtx[e1*V + e2].second = cp;
    }

    // printing values to check
    printf("Initial adj mtx\n");
    for(int i=0;i<cpu_graph->V;i++)
    {
        for(int j=0;j<cpu_graph->V;j++)
        {
            printf("%d/%d ",cpu_graph->adj_mtx[i*cpu_graph->V + j].first,cpu_graph->adj_mtx[i*cpu_graph->V + j].second);
        }
        printf("\n");
    }
}

void preflow(Graph *cpu_graph, int source)
{
    for(int i = 0; i < cpu_graph->V; i++)
    {
        cpu_graph->height[i] = 0;
        cpu_graph->excess_flow[i] = 0;
        
        if(cpu_graph->adj_mtx[source*cpu_graph->V + i].second > 0)
        {
            cpu_graph->adj_mtx[source*cpu_graph->V + i].first = 0;
            cpu_graph->excess_flow[i] += cpu_graph->adj_mtx[source*cpu_graph->V + i].second;
            cpu_graph->excess_total += cpu_graph->excess_flow[i];
        }        
    }

    cpu_graph->height[source] = cpu_graph->V;   
    
    // printing values to check
    for(int i=0;i<cpu_graph->V;i++)
            {
                printf("%d ",cpu_graph->height[i]);
            }
            printf("\nExcess flow :\n");
            for(int i=0;i<cpu_graph->V;i++)
            {
                printf("%d ",cpu_graph->excess_flow[i]);
            }
    printf("\nAdj mtx after preflow\n");
    for(int i=0;i<cpu_graph->V;i++)
    {
        for(int j=0;j<cpu_graph->V;j++)
        {
            printf("%d/%d ",cpu_graph->adj_mtx[i*cpu_graph->V + j].first,cpu_graph->adj_mtx[i*cpu_graph->V + j].second);
        }
        printf("\n");
    }

}

__global__ void push_relabel_kernel(Graph *gpu_graph)
{
    int cycle = KERNEL_CYCLES;
    unsigned int u = (blockIdx.x*blockDim.x) + threadIdx.x;
    //printf("%d \n",u);
    int e1,h1,h2,v,v1,d;

    while(cycle > 0)
    {
        if(gpu_graph->excess_flow[u] > 0 && gpu_graph->height[u] < gpu_graph->V)
        {
            e1 = gpu_graph->excess_flow[u];
            h1 = INF;

            for(int i = 0; i < gpu_graph->V; i++)
            {
                int ind = (gpu_graph->V*u) + i;

                if(gpu_graph->adj_mtx[ind].first > 0)
                {
                    v = i;
                    h2 = gpu_graph->height[i];

                    if(h2 < h1)
                    {
                        v1 = v;
                        h1 = h2;
                    }
                }
            }

            if(gpu_graph->height[u] > h1)
            {
                d = std::min(e1,(gpu_graph->adj_mtx[u*gpu_graph->V + v].first));
                atomicAdd(&(gpu_graph->adj_mtx[v1*gpu_graph->V + u].first), d);
                atomicSub(&(gpu_graph->adj_mtx[u*gpu_graph->V + v1].first), d);
                atomicAdd(&(gpu_graph->excess_flow[v1]), d);
                atomicSub(&(gpu_graph->excess_flow[u]), d);
            }
            else
            {
                gpu_graph->height[u] = h1 + 1;
            }

            //
            // printing values to check
            //
            if(u == 1)
            {
            printf("After iteration %d\n",gpu_graph->V - cycle);
            printf("Height :\n");
            for(int i=0;i<gpu_graph->V;i++)
            {
                printf("%d ",gpu_graph->height[i]);
            }
            printf("\nExcess flow :\n");
            for(int i=0;i<gpu_graph->V;i++)
            {
                printf("%d ",gpu_graph->excess_flow[i]);
            }
            printf("\nAdj mtx after iteration %d :\n",KERNEL_CYCLES - cycle);
            for(int i=0;i<gpu_graph->V;i++)
            {
                for(int j=0;j<gpu_graph->V;j++)
                {
                    printf("%d/%d ",gpu_graph->adj_mtx[i*gpu_graph->V + j].first,gpu_graph->adj_mtx[i*gpu_graph->V + j].second);
                }
                printf("\n");
            }
            }
            //
            //
            //
        }

        //__syncthreads();

        cycle = cycle - 1;

    }

}

void global_relabel(Graph *cpu_graph, int source, int sink)
{
    for(int u = 0; u < cpu_graph->V; u++)
    {
        for(int v = 0; v < cpu_graph->V; v++)
        {
            int ind = (u*cpu_graph->V) + v;
            int ind_trans = (v*cpu_graph->V) + u;

            if((cpu_graph->adj_mtx[ind].second - cpu_graph->adj_mtx[ind].first) > 0)
            {
                if(cpu_graph->height[u] > cpu_graph->height[v] + 1)
                {
                    cpu_graph->excess_flow[u] -= (cpu_graph->adj_mtx[ind].first);
                    cpu_graph->excess_flow[v] += (cpu_graph->adj_mtx[ind].first);
                    cpu_graph->adj_mtx[ind_trans].first += (cpu_graph->adj_mtx[ind].first);
                    cpu_graph->adj_mtx[ind].first = 0; 
                }

            }

        }

        bool mark[cpu_graph->V];
        memset(mark,false,sizeof(mark));

        // bfs routine
        std::list<int> queue;
        int x = source;
        int level = cpu_graph->V;

        mark[source] = true;
        queue.push_back(source);

        while(!queue.empty())
        {
            x = queue.front();
            cpu_graph->height[x] = level;
            queue.pop_front();

            for(int i = 0; i < cpu_graph->V; i++)
            {
                if(cpu_graph->adj_mtx[x*cpu_graph->V + i].first > 0 && !mark[i])
                {
                    mark[i] = true;
                    cpu_graph->height[i] = level - 1;
                }
            }
            level -= 1;
        }

        for(int i = 0; i < cpu_graph->V; i++)
        {
            if(mark[i] == false)
            {
                mark[i] = true;
                cpu_graph->excess_total -= cpu_graph->excess_flow[i];
            }
        }

    }
    
}

void push_relabel(Graph *cpu_graph, Graph *gpu_graph, int source, int sink, int *gpu_height, int *gpu_excess_flow, pii *gpu_adj_mtx)
{
    
    while(cpu_graph->excess_flow[source] + cpu_graph->excess_flow[sink] < cpu_graph->excess_total)
    {
        hipMemcpy(gpu_height,cpu_graph->height,(cpu_graph->V)*sizeof(int),hipMemcpyHostToDevice);

        push_relabel_kernel<<<number_of_blocks_nodes,threads_per_block>>>(gpu_graph);

        hipMemcpy(cpu_graph->adj_mtx,gpu_adj_mtx,cpu_graph->V*cpu_graph->V*sizeof(pii),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_graph->height,gpu_height,cpu_graph->V*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_graph->excess_flow,gpu_excess_flow,cpu_graph->V*sizeof(int),hipMemcpyDeviceToHost);

        //global_relabel(cpu_graph,source,sink);
    }
    printf("maxflow : %d\n",cpu_graph->excess_flow[sink]);
}


int main(int argc, char **argv)
{
    // checking if sufficient arguments are passed in runtime
    if(argc < 5)
    {
        printf("Insufficient number of arguments\n");
        exit(0);
    }

    // reading the arguments passed in CLI
    int V = atoi(argv[1]);
    int E = atoi(argv[2]);
    int source = atoi(argv[3]);
    int sink = atoi(argv[4]);
    
    // initialising Graph variables for host and device
    Graph *cpu_graph,*gpu_graph;
    int *cpu_height,*gpu_height;
    int *cpu_excess_flow,*gpu_excess_flow;
    pii *cpu_adj_mtx,*gpu_adj_mtx;

    // allocating host memory for variables stored on the CPU
    cpu_graph = (Graph*)malloc(sizeof(Graph));
    cpu_height = (int*)malloc(V*sizeof(int));
    cpu_excess_flow = (int*)malloc(V*sizeof(int));
    cpu_adj_mtx = (pii*)malloc(V*V*sizeof(pii));

    // allocating CUDA device global memory for variables stored on the GPU
    hipMalloc((void**)&gpu_graph,sizeof(Graph));
    hipMalloc((void**)&gpu_height,V*sizeof(int));
    hipMalloc((void**)&gpu_excess_flow,V*sizeof(int));
    hipMalloc((void**)&gpu_adj_mtx,V*V*sizeof(pii));

    // Assigning values to the Graph object in the host memory
    cpu_graph->V = V;
    cpu_graph->excess_total = 0;
    cpu_graph->height = cpu_height;
    cpu_graph->excess_flow = cpu_excess_flow;
    cpu_graph->adj_mtx = cpu_adj_mtx;

    // readgraph() - add capacity values to cpu_adj_mtx
    readgraph(cpu_graph,V,E);

    // time start

    // invoking the preflow function 
    preflow(cpu_graph,source);

    // copying the graph from host memory to CUDA device global memory
    hipMemcpy(gpu_graph,cpu_graph,sizeof(Graph),hipMemcpyHostToDevice);
    hipMemcpy(gpu_height,cpu_height,V*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpu_excess_flow,cpu_excess_flow,V*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpu_adj_mtx,cpu_adj_mtx,V*V*sizeof(pii),hipMemcpyHostToDevice);

    // assigning values to pointers of the Graph object in the CUDA device global memory
    hipMemcpy(&(gpu_graph->height),&gpu_height,sizeof(int*),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_graph->excess_flow),&gpu_excess_flow,sizeof(int*),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_graph->adj_mtx),&gpu_adj_mtx,sizeof(pii*),hipMemcpyHostToDevice);

    // invoking the push_relabel host function
    // problem in this function
    push_relabel(cpu_graph,gpu_graph,source,sink,gpu_height,gpu_excess_flow,gpu_adj_mtx);

    printf("Fn over\n");

    // copying the graph from the CUDA device global memory back to host memory
    hipMemcpy(cpu_graph,gpu_graph,sizeof(Graph),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_height,gpu_height,V*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_excess_flow,gpu_excess_flow,V*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_adj_mtx,gpu_adj_mtx,V*V*sizeof(pii),hipMemcpyDeviceToHost);

    // assigning values to pointers of the Graph object in the host memory
    cpu_graph->height = cpu_height;
    cpu_graph->excess_flow = cpu_excess_flow;
    cpu_graph->adj_mtx = cpu_adj_mtx;

    // printing maximum flow of the flow network
    printf("The maximum flow of the flow network is %d\n",cpu_graph->excess_flow[sink]);

    // time end

    // write times to file 

    // free device memory

    // free host memory
}